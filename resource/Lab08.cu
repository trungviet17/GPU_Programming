#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

#define THREADPERBLOCK 256



__global__ void dotProductKernel(int* A, int* B, int* C, int N) {
    __shared__ int sdata[THREADPERBLOCK];


    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = )


}


void runDotProduct(int N) {
    float * h_a, *h_b, *h_c; 
    float *d_a, *d_b, *d_c; z

    size_t size = N * sizeof(float);

    hipHostMalloc((void**)&h_a, size);
    hipHostMalloc((void**)&h_b, size);
    hipHostMalloc((void**)&h_c, size);


    hipHostGetDevicePointer((void**)&d_a, h_a, 0);
    hipHostGetDevicePointer((void**)&d_b, h_b, 0);
    hipHostGetDevicePointer((void**)&d_c, h_c, 0);

    for (int i = 0; i < N; i++) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i);
    }


    int BLOCKPERGRID = (N + THREADPERBLOCK - 1) / THREADPERBLOCK;
    dotProductKernel<<<BLOCKPERGRID, THREADPERBLOCK>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    


}


int main() {
    int N = 1024;
    runDotProduct(N);
    return 0;
}
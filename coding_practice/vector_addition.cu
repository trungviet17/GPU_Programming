#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define N 10 

/*
Vector addition example in CUDA 
Input : Two vectors A and B of size N
Output: Vector C of size N where C[i] = A[i] + B[i]
*/

__global__ void vector_add(int* a, int* b, int* c) {

    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    if (i < N) { 
        c[i] = a[i] + b[i]; 
    }

}


int main() {


    int* h_a, *h_b, *h_c; // Host vectors
    int* d_a, *d_b, *d_c; // Device vectors

    size_t size = N * sizeof(int);
    
    hipHostMalloc((void**)&h_a, size, hipHostMallocDefault);
    hipHostMalloc((void**)&h_b, size, hipHostMallocDefault);
    hipHostMalloc((void**)&h_c, size, hipHostMallocDefault);

    for (int i = 0; i < N; i++) {
        h_a[i] = 1; 
        h_b[i] = 1;
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, size, hipMemcpyHostToDevice);


    int threadPerBlock = 256; 
    int blockPerGrid = (N + threadPerBlock - 1) / threadPerBlock;

    vector_add<<<blockPerGrid, threadPerBlock>>>(d_a, d_b, d_c);
    hipDeviceSynchronize(); 

    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for(int i = 0; i < N; i++) {
        cout << h_c[i] << " "; 
    }
    cout << endl;
}


#include <hip/hip_runtime.h>
#include <iostream> 


__global__ void vectorAdd(int* A, int* B, int* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }

}



void runVectorAdd(int N) {

    std::cout << "Running vector addition with N = " << N << std::endl;

    size_t size = N * sizeof(int);
    int* h_A, *h_B, *h_C;
    int* d_A, *d_B, *d_C;
    

    // 
    hipHostMalloc((void**)&h_A, size, hipHostMallocMapped, hipHostMallocDefault);
    hipHostMalloc((void**)&h_B, size, hipHostMallocMapped, hipHostMallocDefault);
    hipHostMalloc((void**)&h_C, size, hipHostMallocMapped, hipHostMallocDefault);


    hipHostGetDevicePointer((void**)&d_A, (void*)h_A, 0);
    hipHostGetDevicePointer((void**)&d_B, (void*)h_B, 0);
    hipHostGetDevicePointer((void**)&d_C, (void*)h_C, 0);


    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    for (int i = 0; i < N; i++) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            std::cerr << "Error at index " << i << ": " << h_C[i] << " != " << h_A[i] + h_B[i] << std::endl;
            break;
        }
    }
    std::cout << "Vector addition completed successfully!" << std::endl;
    
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipDeviceReset();
    std::cout << "CUDA device reset." << std::endl;

}

int main() {
    int N = 1024 * 1024; 
    runVectorAdd(N);
    return 0;
}

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <tuple>

template<typename T>
__host__ void verifyResult(T *h_a, T *h_b, T *h_c, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int j = 0; j < N; j++) {
      T sum = 0;
      for (int k = 0; k < K; k++) {
        sum += h_a[i * K + k] * h_b[k * N + j];
      }
      //if (i == 0 && j == 0) {
      //  printf("sum: %f, h_c[%d * K + %d]: %f\n", sum, i, j, h_c[i * K + j]);
      //}
      assert(h_c[i * N + j] == sum);
    }
  }
  printf("Correct!\n");
}

template<typename T, size_t BM, size_t BN, size_t BK, size_t TM>
__global__ void gemm_kernel(T* A, T* B, T* C, size_t M, size_t N, size_t K) {
  const uint cRow = blockIdx.y;
  const uint cCol = blockIdx.x;

  // each warp will calculate 32*TM elements, with 32 being the columnar dim.
  const int threadCol = threadIdx.x % BN;
  const int threadRow = threadIdx.x / BN;

  // allocate space for the current blocktile in SMEM
  __shared__ float As[BM * BK];
  __shared__ float Bs[BK * BN];

  // Move blocktile to beginning of A's row and B's column
  A += cRow * BM * K;
  B += cCol * BN;
  C += cRow * BM * N + cCol * BN;

  // todo: adjust this to each thread to load multiple entries and
  // better exploit the cache sizes
  assert(BM * BK == blockDim.x);
  assert(BN * BK == blockDim.x);
  const uint innerColA = threadIdx.x % BK; // warp-level GMEM coalescing
  const uint innerRowA = threadIdx.x / BK;
  const uint innerColB = threadIdx.x % BN; // warp-level GMEM coalescing
  const uint innerRowB = threadIdx.x / BN;

  // allocate thread-local cache for results in registerfile
  float threadResults[TM] = {0.0};

  // outer loop over block tiles
  for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
    // populate the SMEM caches
    As[innerRowA * BK + innerColA] = A[innerRowA * K + innerColA];
    Bs[innerRowB * BN + innerColB] = B[innerRowB * N + innerColB];
    __syncthreads();

    // advance blocktile
    A += BK;
    B += BK * N;

    // calculate per-thread results
    for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
      // we make the dotproduct loop the outside loop, which facilitates
      // reuse of the Bs entry, which we can cache in a tmp var.
      float tmpB = Bs[dotIdx * BN + threadCol];
      for (uint resIdx = 0; resIdx < TM; ++resIdx) {
        threadResults[resIdx] +=
            As[(threadRow * TM + resIdx) * BK + dotIdx] * tmpB;
      }
    }
    __syncthreads();
  }

  // write out the results
  for (uint resIdx = 0; resIdx < TM; ++resIdx) {
    C[(threadRow * TM + resIdx) * N + threadCol] = threadResults[resIdx];
  }
}

template<typename T>
__host__ void copyFromHostToDevice(T* h_a, T* h_b, T* d_a, T* d_b, size_t M, size_t N , size_t K) {
  size_t a_bytes = sizeof(T) * M * K;
  size_t b_bytes = sizeof(T) * K * N;
  hipError_t err = hipMemcpy(d_a, h_a, a_bytes, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy h_a to d_a (error code: %s)", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_b, h_b, b_bytes, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy h_b to d_b (error code: %s)", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

template<typename T, const uint BM, const uint BN, const uint BK, const uint TM>
__host__ void executeKernel(T* d_a, T* d_b, T* d_c, size_t M, size_t N, size_t K) {
  dim3 block((BM * BN) / TM);
  dim3 grid((M + BM - 1) / BM, (N + BN - 1) / BN);
  gemm_kernel<T, BM, BN, BK, TM><<<grid, block>>>(d_a, d_b, d_c, M, N, K);
  hipDeviceSynchronize();
}

template<typename T>
__host__ void copyFromDeviceToHost(T* d_c, T* h_c, size_t M, size_t N) {
  size_t c_bytes = sizeof(T) * M * N;
  hipError_t err = hipMemcpy(h_c, d_c, c_bytes, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy from d_c to h_c (error code: %s)", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

template<typename T>
__host__ void deallocateMemory(T* d_a, T* d_b, T* d_c) {
  hipError_t err = hipFree(d_a);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deallocate d_a (error code: %s)", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipFree(d_b);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deallocate d_b (error code: %s)", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  err = hipFree(d_c);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deallocate d_c (error code: %s)", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__host__ void cleanUpDevice() {
  hipError_t err = hipDeviceReset();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to clean up device (error code: %s)", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__host__ std::tuple<int, int, int> parseCmdLineArgs(int argc, char *argv[]) {
  int M = 1024;
  int N = 1024;
  int K = 1024;

  for (int i = 1; i < argc; i++){
    std::string option(argv[i]);
    std::string value(argv[i+1]);
    i++;
    if (option.compare("-m") == 0) {
      M = std::stoi(value);
    }
    else if (option.compare("-n") == 0) {
      N = std::stoi(value);
    }
    else if (option.compare("-k") == 0) {
      K = std::stoi(value);
    }
  }
  return {M, N, K};
}

int main(int argc, char *argv[]) {
std::tuple<int, int, int>parsedCmdLineArgsTuple = parseCmdLineArgs(argc, argv);
  int M = std::get<0>(parsedCmdLineArgsTuple);
  int N = std::get<1>(parsedCmdLineArgsTuple);
  int K = std::get<2>(parsedCmdLineArgsTuple);
  float* h_a = (float*)malloc(M * K * sizeof(float));
  float* h_b = (float*)malloc(K * N * sizeof(float));
  float* h_c = (float*)malloc(M * N * sizeof(float));

  // initialize
  for (size_t i = 0; i < M; i++) {
    for (size_t j = 0; j < K; j++) {
      h_a[i * K + j] = rand() % 10;
    }
  }

  for (size_t i = 0; i < K; i++) {
    for (size_t j = 0; j < N; j++) {
      h_b[i * N + j] = rand() % 10;
    }
  }

  // allocate memory on device side
  float *d_a, *d_b, *d_c;
  hipMalloc((float **)&d_a, M * K * sizeof(float));
  hipMalloc((float **)&d_b, K * N * sizeof(float));
  hipMalloc((float **)&d_c, M * N * sizeof(float));

  copyFromHostToDevice<float>(h_a, h_b, d_a, d_b, M, N, K);

  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord( start, 0 );

  executeKernel<float, 64, 64, 8, 8>(d_a, d_b, d_c, M, N, K);

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );

  hipEventElapsedTime( &time, start, stop );
  printf("Time taken for GEMM: %f ms\n", time);
  hipEventDestroy( start );
  hipEventDestroy( stop );

  std::cout << "Performance: " << 2LL*M*N*K/(time * 1e-3 * 1e9) << " GFLOP/s\n";

  copyFromDeviceToHost<float>(d_c, h_c, M, N);
  verifyResult<float>(h_a, h_b, h_c, M, N, K);
  deallocateMemory<float>(d_a, d_b, d_c);
  cleanUpDevice();
  return 0;
}